#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#define random(a, b) (rand() % (b - a) + a)
void FillMatrix(float *matrix, int row, int col);
void PrintMatrix(float *A, float *B, float *C, int m, int n, int k);
__global__ void MatrixMulCUDA(const float *A, const float *B, float *C, int m, int n, int k, int ThreadBlockSize)
{
    const int tid = threadIdx.x;
    const int row = tid;
    for (int i = row; i < m; i = i + ThreadBlockSize)
    {
        for (int j = 0; j < k; ++j)
        {
            int temp = 0;
            for (int z = 0; z < n; ++z)
                temp += A[i * n + z] * B[z * k + j];
            C[i * k + j] = temp;
        }
    }
}
int main(int argc, char **argv)
{
    if (argc != 4)
    {
        printf("Wrong Input!\n");
        return 1;
    }
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);
    float *A, *B, *C;
    A = new float[m * n];
    B = new float[n * k];
    C = new float[m * k];
    FillMatrix(A, m, n);
    FillMatrix(B, n, k);
    float elapsedTime;

    float *cuda_A, *cuda_B, *cuda_C;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void **)&cuda_A, sizeof(float) * m * n);
    hipMalloc((void **)&cuda_B, sizeof(float) * n * k);
    hipMalloc((void **)&cuda_C, sizeof(float) * m * k);

    hipMemcpy(cuda_A, A, sizeof(float) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, sizeof(float) * n * k, hipMemcpyHostToDevice);
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                k, //矩阵B的列数
                m, //矩阵A的行数
                n, //矩阵A的列数
                &alpha,
                cuda_B,
                k,
                cuda_A,
                n,
                &beta,
                cuda_C,
                k);

    hipMemcpy(C, cuda_C, sizeof(float) * m * k, hipMemcpyDeviceToHost);
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Calculation time is %.10f ms\n", elapsedTime);
//     PrintMatrix(A, B, C, m, n, k);
    delete[] A;
    delete[] C;
    delete[] B;
    return 0;
}
void FillMatrix(float *matrix, int row, int col)
{
    for (int i = 0; i < row; ++i)
        for (int j = 0; j < col; ++j)
            matrix[i * col + j] = random(0, 9);
}

void PrintMatrix(float *A, float *B, float *C, int m, int n, int k)
{
    printf("Matrix A:\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
            printf("%f ", A[i * n + j]);
        printf("\n");
    }
    printf("Matrix B:\n");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < k; ++j)
            printf("%f ", B[i * k + j]);
        printf("\n");
    }
    printf("Matrix C:\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
            printf("%f ", C[i * k + j]);
        printf("\n");
    }
}