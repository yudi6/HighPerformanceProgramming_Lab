#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define random(a, b) (rand() % (b - a) + a)
#define index(i, j, col) (((i) * (col)) + (j))

void PrintMatrix(float *A, int row, int col);
void FillMatrix(float *matrix, int row, int col, int padding);

__global__ void im2col(float *matrix, int channel_id,
                       int channel, float *matrix_flatten,
                       int height_stride, int width_stride,
                       int matrix_height, int matrix_width,
                       int filter_height, int filter_width,
                       int result_height, int result_width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    //   展平
    if (i < result_height && j < result_width)
        for (int x = 0; x < filter_height; x++)
            for (int y = 0; y < filter_width; y++)
                matrix_flatten[index(index(i, j, result_width), 
                             index(x, y, filter_width) + channel_id * filter_height * filter_width,
                             channel * filter_height * filter_width)]
                = matrix[index(i * height_stride + x, j * width_stride + y, matrix_width)];
    __syncthreads();
}

__global__ void MatrixMulCUDA(float *A, float *B, float *C, int m, int n, int k)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0;
    if (i < m && j < k)
    {
        for (int x = 0; x < n; x++)
            sum += A[i * n + x] * B[x * k + j];
        C[i * k + j] = sum;
    }
}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        printf("Wrong Input!\n");
        return 1;
    }

    int size = atoi(argv[1]);
    int stride = atoi(argv[2]);
    
    int x = atoi(argv[3]);
    int y = atoi(argv[4]);
    dim3 threadsPerBlock(x, y);
    
    int channel = 3;
    float *matrix[channel];
    float *filter[channel];
    float *result;
    
    int matrix_height = size;
    int matrix_width = size;
    
    int filter_height = 3;
    int filter_width = 3;

    int padding = ((((matrix_height - filter_height) / stride + 1) * stride - (matrix_height - filter_height)) % stride) / 2;
    int matrix_size = sizeof(float) * (matrix_height + 2 * padding) * (matrix_width + 2 * padding);
    int result_size = sizeof(float) * ((matrix_height - filter_height + 2 * padding) / stride + 1) * ((matrix_width - filter_width + 2 * padding) / stride + 1);
    int filter_size = sizeof(float) * filter_height * filter_width;

    for (int i = 0; i < channel; i++)
    {
        matrix[i] = (float *)malloc(matrix_size);
        memset(matrix[i], 0, sizeof(matrix[i]));
        FillMatrix(matrix[i], matrix_height, matrix_width, padding);
    }
    for (int i = 0; i < channel; i++)
    {
        filter[i] = (float *)malloc(filter_size);
        for (int j = 0; j < filter_height * filter_width; j++)
            filter[i][j] = j + 1;
    }
    result = (float *)malloc(result_size);
    
    timeval t1, t2;
    gettimeofday(&t1, NULL);
    
    float *cuda_matrix[channel];
    float *cuda_filter[channel];
    float *cuda_result;

    for (int i = 0; i < channel; i++)
    {
        hipMalloc(&cuda_matrix[i], matrix_size);
        hipMemcpy(cuda_matrix[i], matrix[i], matrix_size, hipMemcpyHostToDevice);
    }
    for (int i = 0; i < channel; i++)
    {
        hipMalloc(&cuda_filter[i], filter_size);
        hipMemcpy(cuda_filter[i], filter[i], filter_size, hipMemcpyHostToDevice);
    }
    hipMalloc(&cuda_result, result_size);

    float *matrix_flatten;
    hipMalloc(&matrix_flatten, result_size * filter_height * filter_width * channel);

    int result_height = (matrix_height - filter_height + 2 * padding) / stride + 1;
    int result_width = (matrix_width - filter_width + 2 * padding) / stride + 1;
    
    dim3 numBlocks((result_height % threadsPerBlock.x) ? result_height / threadsPerBlock.x + 1 : result_height / threadsPerBlock.x, (result_width % threadsPerBlock.y) ? result_width / threadsPerBlock.y + 1 : result_width / threadsPerBlock.y);

    for (int i = 0; i < channel; i++)
        im2col<<<numBlocks, threadsPerBlock>>>(cuda_matrix[i], i, channel, matrix_flatten, stride, stride, matrix_height + 2 * padding, matrix_width + 2 * padding, filter_height, filter_width, result_height, result_width);

    float *filter_flatten;
    hipMalloc(&filter_flatten, filter_size * channel);

    for (int i = 0; i < channel; i++)
        hipMemcpy(filter_flatten + i * (filter_height * filter_width), filter[i], filter_size, hipMemcpyHostToDevice);

    dim3 numBlocks1((result_height * result_width % threadsPerBlock.x) ? result_height * result_width / threadsPerBlock.x + 1 : result_height * result_width / threadsPerBlock.x, (filter_height * filter_width % threadsPerBlock.y) ? filter_height * filter_width / threadsPerBlock.y + 1 : filter_height * filter_width / threadsPerBlock.y);

    MatrixMulCUDA<<<numBlocks1, threadsPerBlock>>>(matrix_flatten, filter_flatten, cuda_result, result_height * result_width, filter_height * filter_width * channel, 1);
    gettimeofday(&t2, NULL);
    
    printf("Matrix Size:%d\tStride:%d\n", size, stride);
    printf("Calculation time:%ldms\n", t2.tv_sec * 1000 + t2.tv_usec/1000 - t1.tv_sec * 1000 - t1.tv_usec/1000);
    hipMemcpy(result, cuda_result, result_size, hipMemcpyDeviceToHost);
    
//     for (int i = 0; i < channel; i++)
//     {
//         printf("Matrix after padding of channel %d:\n",i);
//         PrintMatrix(matrix[i], matrix_height + 2 * padding, matrix_width + 2 * padding);
//     }
//     for (int i = 0; i < channel; i++)
//     {
//         printf("Filter of channel %d:\n",i);
//         PrintMatrix(filter[i], filter_height, filter_width);
//     }
//     printf("Result:\n");
//     PrintMatrix(result, ((matrix_height - filter_height + 2 * padding) / stride + 1), ((matrix_width - filter_width + 2 * padding) / stride + 1));
    
    for (int i = 0; i < channel; i++)
        hipFree(cuda_matrix[i]);
    for (int i = 0; i < channel; i++)
        hipFree(cuda_filter[i]);
    hipFree(cuda_result);
    for (int i = 0; i < channel; i++)
        free(matrix[i]);
    for (int i = 0; i < channel; i++)
        free(filter[i]);
    free(result);
}

void FillMatrix(float *matrix, int row, int col, int padding)
{
    for (int i = padding; i < row + padding; i++)
        for (int j = padding; j < col + padding; j++)
            matrix[index(i, j, col + 2 * padding)] = random(0, 9);
}

void PrintMatrix(float *A, int row, int col)
{
    for (int i = 0; i < row; ++i)
    {
        for (int j = 0; j < col; ++j)
            printf("%f ", A[i * col + j]);
        printf("\n");
    }
}